
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    int tid = threadIdx.x;
    printf("Hello world form GPU thread %d!\n", tid);
}

int main(void)
{
    printf("Hello world form CPU!\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    // cudaDeviceSynchronize();
    return 0;
}