
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello world form GPU!\n");
}

int main(void)
{
    printf("Hello world form CPU!\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}